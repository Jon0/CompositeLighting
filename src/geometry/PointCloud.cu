#include "hip/hip_runtime.h"
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void cloud_intersect( int primIdx ) {
  int3 v_idx = vindex_buffer[primIdx];

  float3 p0 = vertex_buffer[ v_idx.x ];
  float3 p1 = vertex_buffer[ v_idx.y ];
  float3 p2 = vertex_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      // Calculate normals and tex coords
      float3 geo_n = normalize( n );
      int3 n_idx = nindex_buffer[ primIdx ];
      if ( normal_buffer.size() == 0 || n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0 ) {
        shading_normal = geo_n;
      } else {
        float3 n0 = normal_buffer[ n_idx.x ];
        float3 n1 = normal_buffer[ n_idx.y ];
        float3 n2 = normal_buffer[ n_idx.z ];
        shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
      }
      geometric_normal = geo_n;

      int3 t_idx = tindex_buffer[ primIdx ];
      if ( texcoord_buffer.size() == 0 || t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0 ) {
        texcoord = make_float3( 0.0f, 0.0f, 0.0f );
      } else {

        float2 t0 = texcoord_buffer[ t_idx.x ];
        float2 t1 = texcoord_buffer[ t_idx.y ];
        float2 t2 = texcoord_buffer[ t_idx.z ];
        texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
      }

      refine_and_offset_hitpoint( ray.origin + t*ray.direction, ray.direction,
                                  geo_n, p0,
                                  back_hit_point, front_hit_point );

      rtReportIntersection( material_buffer[primIdx] );
    }
  }
}
