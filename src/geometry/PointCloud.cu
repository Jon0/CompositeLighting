#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

rtBuffer<float3> vertex_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void cloud_intersect( int primIdx ) {
  float3 p = vertex_buffer[ primIdx ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p, p, p, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {
      rtReportIntersection( 0 );
    }
  }
}
