#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

static __device__ inline float3 perp(float3 p, float a, float b) {
	//return make_float3(a, b, 0);

	return make_float3(p.y * a - p.z * b, - p.x * a, p.x * b);
}

RT_PROGRAM void cloud_intersect(int primIdx) {
	float3 p = vertex_buffer[primIdx];
	float3 nrm = normal_buffer[primIdx];

	float3 p0 = p + perp(nrm, 0.0, 0.5);
	float3 p1 = p + perp(nrm, 0.4, -0.2);
	float3 p2 = p + perp(nrm, -0.4, -0.2);

	// Intersect ray with triangle
	float3 n;
	float t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {
			float3 geo_n = normalize(n);
			geometric_normal = geo_n;
			shading_normal = geo_n;
			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void cloud_bounds (int primIdx, float result[6]) {
	float3 p = vertex_buffer[primIdx];

	optix::Aabb* aabb = (optix::Aabb*) result;

	aabb->m_min = p - make_float3(0.5f);
	aabb->m_max = p + make_float3(0.5f);
}
