#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "helpers.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace {
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	int inside;
	int outline;	// outline mode = 1
};

struct PerRayData_pathtrace_shadow {
	bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      local_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<float4, 2>              output_buffer_empty;	// photo
rtBuffer<float4, 2>              output_buffer_local;
rtBuffer<float4, 2>              output_buffer_all;
rtBuffer<float4, 2>              output_buffer_virt_out;

// use differential rendering
rtDeclareVariable(unsigned int,  display_mode, , );

// Lighting
rtDeclareVariable(float,        lightmap_y_rot, , );

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

// For miss program
rtDeclareVariable(float3,       bg_color, , );

static __device__ inline float3 powf(float3 a, float exp) {
	return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

static __device__ inline float toLinear(float a) {
	if (a < 0.04045) {
		return a / 12.92;
	}
	else {
		return powf((a + 0.055) / 1.055, 2.4);
	}
}

static __device__ inline float toSRGB(float a) {
	if (a < 0.0031308) {
		return a * 12.92;
	}
	else {
		return 1.055*powf(a, 0.416) - 0.055;
	}
}

static __device__ float3 getRay(rtObject geometry, int outline) {
	  size_t2 screen = output_buffer.size();

	  float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
	  float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	  float2 jitter_scale = inv_screen / sqrt_num_samples;
	  unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	  float3 result = make_float3(0.0f);

	  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
	  do {
	    unsigned int x = samples_per_pixel%sqrt_num_samples;
	    unsigned int y = samples_per_pixel/sqrt_num_samples;
	    float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
	    float2 d = pixel + jitter*jitter_scale;
	    float3 ray_origin = eye;
	    float3 ray_direction = normalize(d.x*U + d.y*V + W);

	    PerRayData_pathtrace prd;
	    prd.result = make_float3(0.f);
	    prd.attenuation = make_float3(1.f);
	    prd.countEmitted = true;
	    prd.done = false;
	    prd.inside = false;
	    prd.seed = seed;
	    prd.depth = 0;
	    prd.outline = outline;

	    for(;;) {
	    	// eye ray
	      Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
	      rtTrace(geometry, ray, prd);
	      prd.result += prd.radiance * prd.attenuation;

	      if(prd.done) {
	        break;
	      }

	      // RR
	      prd.depth++;
	      if(prd.depth >= 0){
	        float pcont = fmaxf(prd.attenuation);
	        if(rnd(prd.seed) >= pcont)
	          break;
	        prd.attenuation *= pcont;
	      }

	      ray_origin = prd.origin;
	      ray_direction = prd.direction;
	    }

	    result += prd.result;
	    seed = prd.seed;
	  } while (--samples_per_pixel);

		float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
		pixel_color.x = toSRGB(pixel_color.x);
		pixel_color.y = toSRGB(pixel_color.y);
		pixel_color.z = toSRGB(pixel_color.z);

	return pixel_color;
}

static __device__ float4 getDifferential() {

	// red for virtual geometry
	float geomWeight = output_buffer_virt_out[launch_index].x;

	// green for local geometry
	float localWeight = output_buffer_virt_out[launch_index].y;

	float nongeomWeight = 1.0f - geomWeight;
	float nonlocalWeight = 1.0f - localWeight;

	float4 out = make_float4(0.0f);
	out += geomWeight * output_buffer_all[launch_index];
	out += nonlocalWeight * nongeomWeight * output_buffer_empty[launch_index];

	// local * m = geom
	// +0.01f to avoid divide by zero
	float4 m0 = (output_buffer_all[launch_index]+0.01f) / (output_buffer_local[launch_index]+0.01f);
	out += localWeight * nongeomWeight * output_buffer_empty[launch_index] * m0;

	// doesnt work?
	out = clamp(out, 0.0f, 1.0f);

	return out;
}

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------
RT_PROGRAM void pathtrace_camera() {
	float3 pixel_color_local = getRay(local_object, 0);
	float3 pixel_color_all = getRay(top_object, 0);


	if (frame_number > 1) {
		float a = 1.0f / (float) frame_number;
		float b = ((float) frame_number - 1.0f) * a;

		float3 old_color_all = make_float3(output_buffer_all[launch_index]);
		output_buffer_all[launch_index] = make_float4(a * pixel_color_all + b * old_color_all, 0.0f);

		float3 old_color_local = make_float3(output_buffer_local[launch_index]);
		output_buffer_local[launch_index] = make_float4(a * pixel_color_local + b * old_color_local, 0.0f);

		if (frame_number < 50) {
			float3 old_color_out = make_float3(output_buffer_virt_out[launch_index]);
			output_buffer_virt_out[launch_index] = make_float4(a * getRay(top_object, 1) + b * old_color_out, 0.0f);
		}
	} else {
		// reset buffers
		output_buffer_local[launch_index] = make_float4( pixel_color_local, 0.0f );
		output_buffer_all[launch_index] = make_float4( pixel_color_all, 0.0f );
		output_buffer_virt_out[launch_index] = make_float4( getRay(top_object, 1), 0.0f );
	}

	// final output
	if (display_mode == 1) {
		output_buffer[launch_index] = output_buffer_all[launch_index];
	}
	else if (display_mode == 2) {
		output_buffer[launch_index] = output_buffer_local[launch_index];
	}
	else if (display_mode == 3) {
		output_buffer[launch_index] = output_buffer_virt_out[launch_index];
	}
	else if (display_mode == 4) {
		output_buffer[launch_index] = output_buffer_empty[launch_index];
	}
	else {
		output_buffer[launch_index] = getDifferential();
	}
}

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
  current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(0.f);
  current_prd.done = true;
}

rtDeclareVariable(float3,        diffuse_color, , );
rtDeclareVariable(float3,        outline_color, , );

RT_PROGRAM void diffuse_outline()
{

  current_prd.attenuation = make_float3(1.0f, 0.0f, 0.0f); // red
  current_prd.countEmitted = false;

  float3 result = make_float3(1.0f, 0.0f, 0.0f);

  current_prd.radiance = result;
  current_prd.done = true;
}

RT_PROGRAM void diffuse() {

	if (current_prd.outline == 1) {
		current_prd.countEmitted = false;
		current_prd.radiance = outline_color;
		current_prd.done = true;
		return;
	}
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;

	// normal distribution of outgoing rays with variance of 0.5
	//float r = sqrtf(0.5 * -2 * logf ( rnd(current_prd.seed) ) ) * cosf(2*M_PIf*rnd(current_prd.seed));

	//float ref = rnd(current_prd.seed);

	//if (ref < 0.0) {
		//float3 R = reflect( current_prd.direction, ffnormal );
		//current_prd.direction = R;
	//}
	//else {
		float z1 = rnd(current_prd.seed); // 0.5 + r/2;
		float z2 = rnd(current_prd.seed);
		float3 p;
		cosine_sample_hemisphere(z1, z2, p);
		float3 v1, v2;
		createONB(ffnormal, v1, v2);
		current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
	//}

	//float3 normal_color = (normalize(world_shading_normal) * 0.5f + 0.5f) * 0.9;
	current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
	current_prd.countEmitted = false;

	// Compute light...
	current_prd.radiance = make_float3(0.0f);
}

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );

RT_PROGRAM void glass_refract() {
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;
	current_prd.origin = hitpoint;
	current_prd.countEmitted = true;
	float iof;
	if (current_prd.inside) {
		// Shoot outgoing ray
		iof = 1.0f / index_of_refraction;
	} else {
		iof = index_of_refraction;
	}
	refract(current_prd.direction, ray.direction, ffnormal, iof);
	//prd.direction = reflect(ray.direction, ffnormal);

	if (current_prd.inside) {
		// Compute Beer's law
		current_prd.attenuation = current_prd.attenuation * powf(glass_color, t_hit);
	}
	current_prd.inside = !current_prd.inside;
	current_prd.radiance = make_float3(0.0f);
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}

//-----------------------------------------------------------------------------
//
//  Miss programs
//
//-----------------------------------------------------------------------------
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void miss() {

	if (current_prd.outline == 1) {
		current_prd.radiance = bg_color;
		current_prd.done = true;
		return;
	}

	// sample the light map
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	float3 emap = make_float3(tex2D(envmap, u + lightmap_y_rot, v));
	//emap = emap + 2*powf(emap, 2.0f) + 4*powf(emap, 3.0f)+ 3*powf(emap, 4.0f) + 2*powf(emap, 5.0f);

	current_prd.radiance = emap;
	current_prd.done = true;
}

RT_PROGRAM void miss2()
{
  current_prd.radiance = bg_color;
  current_prd.done = true;
}

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}
